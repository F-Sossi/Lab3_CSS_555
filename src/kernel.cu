#include "hip/hip_runtime.h"
﻿//---------------------------------------------------------------------------
// kernel.cu
// Author: Frank Sossi
// 
// This program implements vector addition using CUDA. The program will
// use a template function as a reference and then compare the results
// to the CUDA implementation.
// 
// To compile: nvcc -arch=sm_86 -lcublas kernel.cu -o lab3
// To run: ./lab3
// Note: -arch=sm_86 is your compute capability mine is 8.6
//---------------------------------------------------------------------------
#include <iostream>
#include <vector>
#include <random>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include ""
#include "gemv.h"

int main() {
	
	// vectors to hold timing data
	std::vector<long long> execution_w_memory;
	std::vector<long long> execution_wo_memory;


	// Allocate memory for each vector on host
	double* vector = (double*)malloc(n * sizeof(double));
	double* matrix = (double*)malloc(n * n * sizeof(double));
	double* ref_result = (double*)malloc(n * sizeof(double));		
	double* calc_result = (double*)malloc(n * sizeof(double));

#ifdef TESTPARAM

	// fill vector with 2's
	for (int i = 0; i < n; i++) {
		vector[i] = 2;
	}

	// fill matrix with 2's
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			matrix[i * n + j] = 2;
		}
	}

#endif

#ifdef REALDATA

	// random number generator
	std::random_device rd;
	std::mt19937 gen(rd());
	std::uniform_real_distribution<> dis(0, 2);

	// fill vector with random numbers
	for (int i = 0; i < n; i++) {
		vector[i] = dis(gen);
	}

	// fill matrix with random numbers
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			matrix[i * n + j] = dis(gen);
		}
	}

#endif

#ifdef DEBUGINPUT

	// print vector
	std::cout << "Vector" << std::endl;
	for (int i = 0; i < n; i++) {
		std::cout << vector[i] << " ";
	}
	std::cout << std::endl;

	// print matrix
	std::cout << "Matrix" << std::endl;
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			std::cout << matrix[i * n + j] << " ";
		}
		std::cout << std::endl;
	}

#endif

#ifdef REFERENCE

	// allocate pointers to GPU memory
	double* device_vector = nullptr;
	double* device_matrix = nullptr;
	double* device_result = nullptr;

	hipMalloc((void**)&device_vector, n * sizeof(double));
	hipMalloc((void**)&device_matrix, n * n * sizeof(double));
	hipMalloc((void**)&device_result, n * sizeof(double));

	// Copy input data to GPU memory
	hipMemcpy(device_vector, vector, n * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(device_matrix, matrix, n * n * sizeof(double), hipMemcpyHostToDevice);

	// Create a handle for cuBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// Perform the vector-matrix multiplication using cuBLAS
	// Perform the matrix-vector multiplication using cuBLAS
	double alpha = 1.0;
	double beta = 0.0;
	hipblasDgemv(handle, HIPBLAS_OP_N, n, n, &alpha, device_matrix, n, device_vector, 1, &beta, device_result, 1);


	// Copy the result from GPU memory to host memory
	hipMemcpy(ref_result, device_result, n * sizeof(double), hipMemcpyDeviceToHost);

	// Destroy the cuBLAS handle
	hipblasDestroy(handle);

	hipFree(device_vector);
	hipFree(device_matrix);
	hipFree(device_result);
		
#endif

#ifdef PART1

	// allocate pointers to GPU memory
	double* device_vector2 = nullptr;
	double* device_matrix2 = nullptr;
	double* device_result2 = nullptr;

	hipMalloc((void**)&device_vector2, n * sizeof(double));
	hipMalloc((void**)&device_matrix2, n * n * sizeof(double));
	hipMalloc((void**)&device_result2, n * sizeof(double));

	// Copy input data to GPU memory
	hipMemcpy(device_vector2, vector, n * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(device_matrix2, matrix, n * n * sizeof(double), hipMemcpyHostToDevice);


	const int num_blocks = (n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK;
	//const int max_blocks = 32767;
	const int blocks = std::min(num_blocks, max_blocks);
	dim3 grid(blocks, 1, 1);
	dim3 block(THREAD_PER_BLOCK, 1, 1);
	gemv_kernel_part1<<<grid, block>>>(device_matrix2, device_vector2, device_result2, n, n);


	hipDeviceSynchronize();
	// Copy the result from GPU memory to host memory
	hipMemcpy(calc_result, device_result2, n * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(device_vector2);
	hipFree(device_matrix2);
	hipFree(device_result2);

#endif

#ifdef PART2

	// allocate pointers to GPU memory
	double* device_vector2 = nullptr;
	double* device_matrix2 = nullptr;
	double* device_result2 = nullptr;

	hipMalloc((void**)&device_vector2, n * sizeof(double));
	hipMalloc((void**)&device_matrix2, n * n * sizeof(double));
	hipMalloc((void**)&device_result2, n * sizeof(double));

	// Copy input data to GPU memory
	hipMemcpy(device_vector2, vector, n * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(device_matrix2, matrix, n * n * sizeof(double), hipMemcpyHostToDevice);


	const int num_blocks = (n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK;
	//const int max_blocks = 32767;
	const int blocks = std::min(num_blocks, max_blocks);
	//const int block_size = BLOCK_SIZE;

	dim3 grid(blocks, 1, 1);
	dim3 block(THREAD_PER_BLOCK, 1, 1);

	gemv_part2_ver1<<<grid, block>>>(device_matrix2, device_vector2, device_result2, n, n);



	hipDeviceSynchronize();
	// Copy the result from GPU memory to host memory
	hipMemcpy(calc_result, device_result2, n * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(device_vector2);
	hipFree(device_matrix2);
	hipFree(device_result2);

#endif

#ifdef PART3

	// allocate pointers to GPU memory
	double* device_vector2 = nullptr;
	double* device_matrix2 = nullptr;
	double* device_result2 = nullptr;

	hipMalloc((void**)&device_vector2, n * sizeof(double));
	hipMalloc((void**)&device_matrix2, n * n * sizeof(double));
	hipMalloc((void**)&device_result2, n * sizeof(double));

	// Copy input data to GPU memory
	hipMemcpy(device_vector2, vector, n * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(device_matrix2, matrix, n * n * sizeof(double), hipMemcpyHostToDevice);


	const int num_blocks = (n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK;
	//const int max_blocks = 32767;
	const int blocks = std::min(num_blocks, max_blocks);
	dim3 grid(blocks, 1, 1);
	dim3 block(THREAD_PER_BLOCK, 1, 1);
	gemv_kernel_part3<<<grid, block>>>(device_matrix2, device_vector2, device_result2, n, n);


	hipDeviceSynchronize();
	// Copy the result from GPU memory to host memory
	hipMemcpy(calc_result, device_result2, n * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(device_vector2);
	hipFree(device_matrix2);
	hipFree(device_result2);

#endif

#ifdef DEBUG

	// print reference result
	std::cout << "Reference Result" << std::endl;
	for (int i = 0; i < n; i++) {
		std::cout << ref_result[i] << " ";
	}
	std::cout << std::endl;

	// print calculated result
	std::cout << "Calculated Result" << std::endl;
	for (int i = 0; i < n; i++) {
		std::cout << calc_result[i] << " ";
	}
	std::cout << std::endl;

#endif

	free(vector);
	free(matrix);
	free(ref_result);
	free(calc_result);
    
}
