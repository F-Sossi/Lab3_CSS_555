#include "hip/hip_runtime.h"
﻿//---------------------------------------------------------------------------
// kernel.cu
// Author: Frank Sossi
// 
// This program implements vector addition using CUDA. The program will
// use a template function as a reference and then compare the results
// to the CUDA implementation.
// 
// To compile: nvcc -arch=sm_86 -lcublas kernel.cu -o lab3
// To run: ./lab3
// Note: -arch=sm_86 is your compute capability mine is 8.6
//---------------------------------------------------------------------------
#include <iostream>
#include <sstream>
#include <string>
#include <limits>
#include <vector>
#include <random>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include ""
#include "gemv.h"


int main() {

	int n ={};
	int THREAD_PER_BLOCK = {};
	std::string want_reference;
	int part_to_run;
    bool quit = false;

	while (!quit) {
		std::cout << "Enter the value of n (less than " << MAX_NUM << "), or 'q' to quit: ";
		std::string input;
		std::cin >> input;
		if (input == "q") {
		quit = true;
		continue;
		}
		while (!(std::stringstream(input) >> n) || n >= MAX_NUM) {
			std::cout << "Invalid input. Please enter the value of n (less than " << MAX_NUM << "), or 'q' to quit: ";
			std::cin.clear();
			std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
			std::cin >> input;
		if (input == "q") {
			quit = true;
			break;
		}
		}
		if (quit) {
		break;
		}

		std::cout << "Enter the number of threads per block, or 'q' to quit: ";
		std::cin >> input;
		if (input == "q") {
		quit = true;
		continue;
		}
		while (!(std::stringstream(input) >> THREAD_PER_BLOCK)) {
			std::cout << "Invalid input. Please enter the number of threads per block, or 'q' to quit: ";
			std::cin.clear();
			std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
			std::cin >> input;
		if (input == "q") {
			quit = true;
			break;
		}
		}
		if (quit) {
		break;
		}

		std::cout << "Do you want to check reference? (y/n), or 'q' to quit: ";
		std::cin >> input;
		if (input == "q") {
		quit = true;
		continue;
		}
		while (input != "y" && input != "n") {
			std::cout << "Invalid input. Please enter 'y' or 'n' for whether you want to check reference, or 'q' to quit: ";
			std::cin.clear();
			std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
			std::cin >> input;
		if (input == "q") {
			quit = true;
			break;
		}
		}
		if (quit) {
		break;
		}
		want_reference = input;

		std::cout << "Which part do you want to run? (1/2/3), or 'q' to quit: ";
		std::cin >> input;
		if (input == "q") {
		quit = true;
		continue;
		}
		while (!(std::stringstream(input) >> part_to_run) || part_to_run < 1 || part_to_run > 3) {
		std::cout << "Invalid input. Please enter 1, 2, or 3 for which part to run, or 'q' to quit: ";
		std::cin.clear();
		std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
		std::cin >> input;
		if (input == "q") {
			quit = true;
			break;
		}
		}
		if (quit) {
		break;
		}
		
		// vectors to hold timing data
		std::vector<long long> execution_w_memory;
		std::vector<long long> execution_wo_memory;


		// Allocate memory for each vector on host
		double* vector = (double*)malloc(n * sizeof(double));
		double* matrix = (double*)malloc(n * n * sizeof(double));
		double* ref_result = (double*)malloc(n * sizeof(double));		
		double* calc_result = (double*)malloc(n * sizeof(double));

	#ifdef TESTPARAM

		// fill vector with 2's
		for (int i = 0; i < n; i++) {
			vector[i] = 2;
		}

		// fill matrix with 2's
		for (int i = 0; i < n; i++) {
			for (int j = 0; j < n; j++) {
				matrix[i * n + j] = 2;
			}
		}

	#endif

	#ifdef REALDATA

		// random number generator
		std::random_device rd;
		std::mt19937 gen(rd());
		std::uniform_real_distribution<> dis(0, 2);

		// fill vector with random numbers
		for (int i = 0; i < n; i++) {
			vector[i] = dis(gen);
		}

		// fill matrix with random numbers
		for (int i = 0; i < n; i++) {
			for (int j = 0; j < n; j++) {
				matrix[i * n + j] = dis(gen);
			}
		}

	#endif

	#ifdef DEBUGINPUT

		// print vector
		std::cout << "Vector" << std::endl;
		for (int i = 0; i < n; i++) {
			std::cout << vector[i] << " ";
		}
		std::cout << std::endl;

		// print matrix
		std::cout << "Matrix" << std::endl;
		for (int i = 0; i < n; i++) {
			for (int j = 0; j < n; j++) {
				std::cout << matrix[i * n + j] << " ";
			}
			std::cout << std::endl;
		}

	#endif




		// allocate pointers to GPU memory
		double* device_vector2 = nullptr;
		double* device_matrix2 = nullptr;
		double* device_result2 = nullptr;

		// get time before allocating memory on GPU
		auto w_memory_start = get_time();

		hipMalloc((void**)&device_vector2, n * sizeof(double));
		hipMalloc((void**)&device_matrix2, n * n * sizeof(double));
		hipMalloc((void**)&device_result2, n * sizeof(double));

		// Copy input data to GPU memory
		hipMemcpy(device_vector2, vector, n * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(device_matrix2, matrix, n * n * sizeof(double), hipMemcpyHostToDevice);


		const int num_blocks = (n + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK;
		//const int max_blocks = 32767;
		const int blocks = std::min(num_blocks, max_blocks);
		dim3 grid(blocks, 1, 1);
		dim3 block(THREAD_PER_BLOCK, 1, 1);

		auto wo_memory_start = get_time();

		switch (part_to_run) {
			case 1:
			gemv_kernel_part1<<<grid, block>>>(device_matrix2, device_vector2, device_result2, n, n);
			break;
			case 2:
				gemv_part2_ver2<<<grid, block>>>(device_matrix2, device_vector2, device_result2, n, n);
			break;
			case 3:
			gemv_kernel_part3<<<grid, block>>>(device_matrix2, device_vector2, device_result2, n, n);
			break;
			default:
			std::cerr << "Invalid part to run" << std::endl;
		return 1;
		}

		auto wo_memory_end = get_time();

		hipError_t err = hipDeviceSynchronize();
		if (err != hipSuccess) {
			printf("Kernel launch failed with error code %d: %s\n", err, hipGetErrorString(err));
		}

		err = hipGetLastError();
		if (err != hipSuccess) {
			printf("Kernel encountered an error: %d: %s\n", err, hipGetErrorString(err));
		}
		
		// Copy the result from GPU memory to host memory
		hipMemcpy(calc_result, device_result2, n * sizeof(double), hipMemcpyDeviceToHost);

		hipFree(device_vector2);
		hipFree(device_matrix2);
		hipFree(device_result2);

		auto w_memory_end = get_time();

		// calculate time for memory allocation
		auto w_memory_time = std::chrono::duration_cast<std::chrono::nanoseconds>(w_memory_end - w_memory_start).count();

		// print time for memory allocation
		std::cout << "Time with memory allocation: " << w_memory_time << std::endl;

		// calculate time without memory allocation
		auto wo_memory_time = std::chrono::duration_cast<std::chrono::nanoseconds>(wo_memory_end - wo_memory_start).count();

		// print time without memory allocation
		std::cout << "Time without memory allocation: " << wo_memory_time << std::endl;



	#ifdef DEBUG

		// print reference result
		std::cout << "Reference Result" << std::endl;
		for (int i = 0; i < n; i++) {
			std::cout << ref_result[i] << " ";
		}
		std::cout << std::endl;

		// print calculated result
		std::cout << "Calculated Result" << std::endl;
		for (int i = 0; i < n; i++) {
			std::cout << calc_result[i] << " ";
		}
		std::cout << std::endl;

	#endif



		if (want_reference == "y") {
			// code to perform reference check


			// allocate pointers to GPU memory
			double* device_vector = nullptr;
			double* device_matrix = nullptr;
			double* device_result = nullptr;

			hipMalloc((void**)&device_vector, n * sizeof(double));
			hipMalloc((void**)&device_matrix, n * n * sizeof(double));
			hipMalloc((void**)&device_result, n * sizeof(double));

			// Copy input data to GPU memory
			hipMemcpy(device_vector, vector, n * sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(device_matrix, matrix, n * n * sizeof(double), hipMemcpyHostToDevice);

			// Create a handle for cuBLAS
			hipblasHandle_t handle;
			hipblasCreate(&handle);

			// Perform the vector-matrix multiplication using cuBLAS
			// Perform the matrix-vector multiplication using cuBLAS
			double alpha = 1.0;
			double beta = 0.0;
			hipblasDgemv(handle, HIPBLAS_OP_N, n, n, &alpha, device_matrix, n, device_vector, 1, &beta, device_result, 1);


			// Copy the result from GPU memory to host memory
			hipMemcpy(ref_result, device_result, n * sizeof(double), hipMemcpyDeviceToHost);

			// Destroy the cuBLAS handle
			hipblasDestroy(handle);

			hipFree(device_vector);
			hipFree(device_matrix);
			hipFree(device_result);

			// verify the result
			double error = 0.0;
			for (int i = 0; i < n; i++) {
				error += (ref_result[i] - calc_result[i]);
			}
			// find average error
			error /= n;
			std::cout << "Average Error: " << error << std::endl;
		}


		free(vector);
		free(matrix);
		free(ref_result);
		free(calc_result);
	}
 
	return 0;
}
