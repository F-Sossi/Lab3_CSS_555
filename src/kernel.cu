﻿//---------------------------------------------------------------------------
// kernel.cu
// Author: Frank Sossi
// 
// This program implements vector addition using CUDA. The program will
// use a template function as a reference and then compare the results
// to the CUDA implementation.
// 
// To compile: nvcc -arch=sm_86 -lcublas kernel.cu -o lab3
// To run: ./lab3
// Note: -arch=sm_86 is your compute capability mine is 8.6
//---------------------------------------------------------------------------
#include <iostream>
#include <vector>
#include <chrono>
#include <random>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cmath>
#include ""
#include "gemv.h"

// Block and thread count to run different configurations when testing
//   different block sizes use add_bloc() when testing different 
//   thread counts use add_thread() add this to check added

#define REFERENCE
#define PART1
#define DEBUG

// Threads 8 64 128 512 1024 
//constexpr auto THREAD_PER_BLOCK = 128;

// Size of the vector
constexpr int n = 10;


int main() {
	
	// vectors to hold timing data
	std::vector<long long> execution_w_memory;
	std::vector<long long> execution_wo_memory;


	// Allocate memory for each vector on host
	double* vector = (double*)malloc(n * sizeof(double));
	double* matrix = (double*)malloc(n * n * sizeof(double));
	double* ref_result = (double*)malloc(n * sizeof(double));		
	double* calc_result = (double*)malloc(n * sizeof(double));		

	// fill vector with 2's
	for (int i = 0; i < n; i++) {
		vector[i] = 2;
	}

	// fill matrix with 2's
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			matrix[i * n + j] = 2;
		}
	}

#ifdef DEBUG

	// print vector
	std::cout << "Vector" << std::endl;
	for (int i = 0; i < n; i++) {
		std::cout << vector[i] << " ";
	}
	std::cout << std::endl;

	// print matrix
	std::cout << "Matrix" << std::endl;
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			std::cout << matrix[i * n + j] << " ";
		}
		std::cout << std::endl;
	}

#endif

#ifdef REFERENCE

	// allocate pointers to GPU memory
	double* device_vector = nullptr;
	double* device_matrix = nullptr;
	double* device_result = nullptr;

	hipMalloc((void**)&device_vector, n * sizeof(double));
	hipMalloc((void**)&device_matrix, n * n * sizeof(double));
	hipMalloc((void**)&device_result, n * n * sizeof(double));

	// Copy input data to GPU memory
	hipMemcpy(device_vector, vector, n * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(device_matrix, matrix, n * n * sizeof(double), hipMemcpyHostToDevice);

	// Create a handle for cuBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// Perform the vector-matrix multiplication using cuBLAS
	// Perform the matrix-vector multiplication using cuBLAS
	double alpha = 1.0;
	double beta = 0.0;
	hipblasDgemv(handle, HIPBLAS_OP_N, n, n, &alpha, device_matrix, n, device_vector, 1, &beta, device_result, 1);


	// Copy the result from GPU memory to host memory
	hipMemcpy(ref_result, device_result, n * sizeof(double), hipMemcpyDeviceToHost);

	// Destroy the cuBLAS handle
	hipblasDestroy(handle);

	hipFree(device_vector);
	hipFree(device_matrix);
	hipFree(device_result);
		
#endif

#ifdef PART1

	// allocate pointers to GPU memory
	double* device_vector2 = nullptr;
	double* device_matrix2 = nullptr;
	double* device_result2 = nullptr;

	hipMalloc((void**)&device_vector2, n * sizeof(double));
	hipMalloc((void**)&device_matrix2, n * n * sizeof(double));
	hipMalloc((void**)&device_result2, n * sizeof(double));

	// Copy input data to GPU memory
	hipMemcpy(device_vector2, vector, n * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(device_matrix2, matrix, n * n * sizeof(double), hipMemcpyHostToDevice);

	// call gemv_kernel
	dim3 grid(n);
	dim3 block(1);
	gemv_kernel<<<grid, block>>>(device_matrix2, device_vector2, device_result2, n, n);

	// Copy the result from GPU memory to host memory
	hipMemcpy(calc_result, device_result2, n * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(device_vector2);
	hipFree(device_matrix2);
	hipFree(device_result2);

#endif

	// print reference result
	std::cout << "Reference Result" << std::endl;
	for (int i = 0; i < n; i++) {
		std::cout << ref_result[i] << " ";
	}
	std::cout << std::endl;

	// print calculated result
	std::cout << "Calculated Result" << std::endl;
	for (int i = 0; i < n; i++) {
		std::cout << calc_result[i] << " ";
	}
	std::cout << std::endl;


	hipFree(device_vector);
	hipFree(device_matrix);
	hipFree(device_result);
		
    
}
